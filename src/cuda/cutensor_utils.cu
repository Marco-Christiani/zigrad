#ifndef __CUTENSOR_UTILS_ZIG__
#define __CUTENSOR_UTILS_ZIG__
#include "decls.h"
#include "cuda_helpers.cu"
#include <cutensor/types.h>

#include <array>
#include <algorithm>
#include <unordered_map>
#include <initializer_list>
#include <typeinfo>
#include <variant>

static const u32 cutensor_alignment = 128; // Alignment of the global-memory device poi32ers (bytes)

hiptensorOperator_t cutensor_op_type(BINARY_OP op) {
  switch (op) {
    case BINARY_OP::ADD: return hiptensorOperator_t::HIPTENSOR_OP_ADD;
    case BINARY_OP::MIN: return hiptensorOperator_t::HIPTENSOR_OP_MIN;
    case BINARY_OP::MAX: return hiptensorOperator_t::HIPTENSOR_OP_MAX;
    case BINARY_OP::MUL: return hiptensorOperator_t::HIPTENSOR_OP_MUL;
    default: 
      SYSTEM_EXIT("Invalid reduce operation");
      return {}; // silence warning
  }
}

hiptensorDataType_t cutensor_data_type(dtype id) {
  switch (id) {
    case SINGLE: return hiptensorDataType_t::HIPTENSOR_R_32F;
    case DOUBLE: return hiptensorDataType_t::HIPTENSOR_R_64F;
    default: 
      SYSTEM_EXIT("Invalid data type");
      return {}; // silence warning
  }
}
  
hiptensorComputeDescriptor_t cutensor_compute_type(dtype id) {
  switch (id) {
    case SINGLE: return HIPTENSOR_COMPUTE_DESC_32F;
    case DOUBLE: return HIPTENSOR_COMPUTE_DESC_64F;
    default: 
      SYSTEM_EXIT("Invalid data type");
      return {}; // silence warning
  }
}

// Taken from: https://stackoverflow.com/questions/20511347/a-good-hash-function-for-a-vector/72073933#72073933
template <class T>
std::size_t __seq_hash(const T* key, std::size_t n) {
  std::size_t seed = n;
  for (std::size_t i = 0; i < n; ++i) {
    T x = key[i];
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    seed ^= x + 0x9e3779b9 + (seed << 6) + (seed >> 2);
  }
  return seed;
}

template<class T, std::size_t N = 8>
struct BoundedArray {
  T data[N];
  std::size_t size = 0;

  using iterator = typename std::array<T, N>::iterator;
  using const_iterator = typename std::array<T, N>::const_iterator;

  BoundedArray() = default;

  template<class U>
  BoundedArray(const U* vals, std::size_t n) : BoundedArray() {
    this->append(vals, n);
  }

  template<class U>
  BoundedArray(const U* vals, std::size_t n, bool reverse) : BoundedArray(vals, n) {
    if (reverse) this->reverse();
  }

  BoundedArray(std::initializer_list<std::size_t> vals) : BoundedArray() {
    this->append(vals);
  }

  std::size_t hash() const {
    return __seq_hash(this->ptr(), this->size);
  }

  iterator begin() {
    return iterator{ &this->data[0] };
  }
  iterator end() {
    return this->begin() + this->size;
  }

  const_iterator begin() const {
    return const_iterator{ &this->data[0] };
  }
  const_iterator end() const {
    return this->begin() + this->size;
  }

  T* ptr() {
    return &this->data[0];
  }

  const T* ptr() const {
    return &this->data[0];
  }

  void reverse() {
    std::reverse(this->begin(), this->end());
  }

  void sort() {
    std::sort(this->begin(), this->end());
  }

  template<class U>
  void append(const U* vals, len_t n) {
    for (len_t i = 0; i < n; ++i) {
      this->append(vals[i]);
    }
  }

  template<class ... Ts>
  void append(std::initializer_list<Ts...> vals) {
    for (auto x: vals) {
      this->append(x);
    }
  }

  template <class U>
  void append(U val) {
    CHECK_INVARIANT((1 + this->size <= N), "Bounded Array Overflow");
    this->data[this->size] = static_cast<T>(val);
    ++this->size;
  }
};

/////////////////////////////////////////////////////////
// Cutensor Data Component Cleanup //////////////////////

void destroy(hiptensorPlan_t ptr) {
  CUTENSOR_ASSERT(hiptensorDestroyPlan(ptr));
}
void destroy(hiptensorPlanPreference_t ptr) {
  CUTENSOR_ASSERT(hiptensorDestroyPlanPreference(ptr));
}
void destroy(hiptensorTensorDescriptor_t ptr) {
  CUTENSOR_ASSERT(hiptensorDestroyTensorDescriptor(ptr));
}
void destroy(hiptensorOperationDescriptor_t ptr) {
  CUTENSOR_ASSERT(hiptensorDestroyOperationDescriptor(ptr));
}

template<class T>
struct ManagedComponent {
  T ptr;
  ManagedComponent(T _ptr = nullptr) : ptr{_ptr} {}
  ManagedComponent(ManagedComponent const&) = delete;
  ManagedComponent& operator=(const ManagedComponent&) = delete;
  ManagedComponent(ManagedComponent&& other) noexcept {
    this->ptr = std::exchange(other.ptr, nullptr);
  };
  ManagedComponent& operator=(ManagedComponent&& other) noexcept {
    this->ptr = std::exchange(other.ptr, nullptr);
  };
  ~ManagedComponent() {
    if (this->ptr) {
      destroy(this->ptr);
    }
  }
};

typedef ManagedComponent<hiptensorPlan_t> ManagedPlan;
typedef ManagedComponent<hiptensorPlanPreference_t> ManagedPlanPreference;
typedef ManagedComponent<hiptensorTensorDescriptor_t> ManagedTensorDescriptor;
typedef ManagedComponent<hiptensorOperationDescriptor_t> ManagedOperationDescriptor;

/////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////

struct ReducePlan {
  ManagedPlan plan;
  ManagedPlanPreference plan_pref;
  ManagedOperationDescriptor op_desc;
  ManagedTensorDescriptor x_desc;
  ManagedTensorDescriptor y_desc;
  len_t scratch_len;
};

struct PermutatePlan {
  ManagedPlan plan;
  ManagedPlanPreference plan_pref;
  ManagedOperationDescriptor op_desc;
  ManagedTensorDescriptor x_desc;
  ManagedTensorDescriptor y_desc;
  len_t scratch_len;
};

struct ContractionPlan {
  ManagedPlan plan;
  ManagedPlanPreference plan_pref;
  ManagedOperationDescriptor op_desc;
  ManagedTensorDescriptor x_desc;
  ManagedTensorDescriptor y_desc;
  ManagedTensorDescriptor z_desc;
  len_t scratch_len;
};

struct BinaryPlan {
  ManagedPlan plan;
  ManagedPlanPreference plan_pref;
  ManagedOperationDescriptor op_desc;
  ManagedTensorDescriptor x_desc;
  ManagedTensorDescriptor y_desc;
  len_t scratch_len;
};

typedef std::variant<
  BinaryPlan,
  ContractionPlan,
  PermutatePlan,
  ReducePlan
> PlanVariant;

typedef std::shared_ptr<PlanVariant> MapPtr;

class PlanManager {
private:
  
  struct MapKey {
    BoundedArray<len_t, 8> data;
  
    struct Hash{
      std::size_t operator()(MapKey key) const { return key.data.hash(); }
    };
    bool operator==(const MapKey& b) const {
      return std::equal(this->data.begin(), this->data.end(), b.data.begin(), b.data.end());
    }
  };

  typedef std::unordered_map<MapKey, MapPtr, MapKey::Hash> MapType;

  MapType map{};

public:

  PlanManager() = default;
  PlanManager(const PlanManager&) = delete;
  PlanManager(PlanManager&&) noexcept = default;

  template<class T>
  MapKey make_key(hiptensorDataType_t dtype, std::initializer_list<std::size_t> vals) {
    MapKey key;
    key.data.append(vals);
    key.data.append(dtype);
    key.data.append(typeid(T).hash_code());
    return key;
  }

  MapType::iterator begin() {
    return this->map.begin();
  }
  MapType::iterator end() {
    return this->map.end();
  }

  template<class T>
  T* find(MapKey const& k) {
    auto itr = this->map.find(k);
    if (itr == this->map.end()) {
      return nullptr;
    }
    return &std::get<T>(*itr->second);
  }

  template<class T>
  void insert(MapKey key, T&& val) {
    this->map.insert(std::make_pair(key, std::make_shared<PlanVariant>(std::move(val))));
  }
};


struct CutensorBackend {
    PlanManager manager;
    hipStream_t stream{nullptr};
    hiptensorHandle_t handle{nullptr};

    static CutensorBackend* unwrap(CutensorWrapper wrapper) {
      return static_cast<CutensorBackend*>(wrapper.ptr);
    }
    static CutensorWrapper wrap(CutensorBackend* backend) {
      return { .ptr = backend };
    }

    CutensorBackend(void* _stream) {
      this->stream = static_cast<hipStream_t>(_stream);
      CUTENSOR_ASSERT(hiptensorCreate(&this->handle));
    }

    ~CutensorBackend() {
      CUTENSOR_ASSERT(hiptensorDestroy(this->handle));
    }
};
  
// binary is used for broadcasting reverses
// not intended to be used with 
//CutensorPlanWrapper get_binary_plan(
//  CutensorWrapper wrapper,
//  dtype id,
//  const len_t* src_dims,
//  const u8* src_syms,
//  len_t src_dims_len,
//  const len_t* dst_dims,
//  const u8* dst_syms,
//  len_t dst_dims_len,
//  BINARY_OP op
//) {
//  CHECK_INVARIANT(0 < src_dims_len, "Zero length dimensions passed to permutate");
//  CHECK_INVARIANT(src_dims_len <= dst_dims_len, "Source dimensions length greater than destination");
//  
//  const auto data_type = cutensor_data_type(id);
//  const auto op_type = cutensor_op_type(op);
//
//  auto ct = CutensorBackend::unwrap(wrapper);
//
//  auto key = ct->manager.make_key<BinaryPlan>(
//    data_type,
//    {
//      __seq_hash(src_dims, src_dims_len),
//      __seq_hash(src_syms, src_dims_len),
//      __seq_hash(dst_dims, src_dims_len),
//      __seq_hash(dst_syms, src_dims_len),
//      static_cast<std::size_t>(op),
//    }
//  );
//
//  if (auto entry = ct->manager.find<PermutatePlan>(key); entry)
//    return { .ptr = entry->plan.ptr, .scratch_len = entry->scratch_len };
//
//  BoundedArray<i64> a_dims(src_dims, src_dims_len, true);
//  BoundedArray<i32> a_syms(src_syms, src_dims_len, true);
//  BoundedArray<i64> b_dims(dst_dims, src_dims_len, true);
//  BoundedArray<i32> b_syms(dst_syms, src_dims_len, true);
//  
//  hiptensorTensorDescriptor_t x_desc;
//  CUTENSOR_ASSERT(hiptensorCreateTensorDescriptor(
//              ct->handle,
//              &x_desc,
//              a_dims.size,
//              a_dims.data,
//              NULL,/*stride*/
//              data_type, cutensor_alignment));
//  
//  hiptensorTensorDescriptor_t y_desc;
//  CUTENSOR_ASSERT(hiptensorCreateTensorDescriptor(
//              ct->handle,
//              &y_desc,
//              b_dims.size,
//              b_dims.data,
//              NULL,/*stride*/
//              data_type, cutensor_alignment));
//  
//  hiptensorOperationDescriptor_t op_desc;
//  CUTENSOR_ASSERT(hiptensorCreateElementwiseBinary(
//              ct->handle, &op_desc,
//              x_desc, a_syms.data, HIPTENSOR_OP_IDENTITY,
//              y_desc, b_syms.data, HIPTENSOR_OP_IDENTITY,
//              y_desc, b_syms.data,
//              op_type, cutensor_compute_type(id)));
//
//  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
//  
//  hiptensorPlanPreference_t plan_pref;
//  CUTENSOR_ASSERT(hiptensorCreatePlanPreference(
//              ct->handle,
//              &plan_pref,
//              algo,
//              HIPTENSOR_JIT_MODE_NONE));
//  
//  
//  hiptensorPlan_t plan;
//  CUTENSOR_ASSERT(hiptensorCreatePlan(
//              ct->handle,
//              &plan,
//              op_desc,
//              plan_pref,
//              0 /* not required */));
//  
//  ct->manager.insert(
//    key,
//    BinaryPlan{
//      .plan = plan,
//      .plan_pref = plan_pref,
//      .x_desc = x_desc,
//      .y_desc = y_desc,
//    }
//  );
//
//  return { .ptr = plan };
//}

// binary is used for broadcasting reverses
// not intended to be used with 
//CutensorPlanWrapper get_reduce_bwds_trinary(
//  CutensorWrapper wrapper,
//  dtype id,
//  const len_t* src_dims,
//  const u8* src_syms,
//  len_t src_dims_len,
//  const len_t* dst_dims,
//  const u8* dst_syms,
//  len_t dst_dims_len,
//  BINARY_OP op
//) {
//  CHECK_INVARIANT(0 < src_dims_len, "Zero length dimensions passed to permutate");
//  CHECK_INVARIANT(src_dims_len <= dst_dims_len, "Source dimensions length greater than destination");
//
//  auto ct = CutensorBackend::unwrap(wrapper);
//  
//  const auto data_type = cutensor_data_type(id);
//  const auto op_type = cutensor_op_type(op);
//
//  auto key = ct->manager.make_key<BinaryPlan>(
//    data_type,
//    {
//      __seq_hash(src_dims, src_dims_len),
//      __seq_hash(src_syms, src_dims_len),
//      __seq_hash(dst_dims, src_dims_len),
//      __seq_hash(dst_syms, src_dims_len),
//      static_cast<std::size_t>(op),
//    }
//  );
//
//  if (auto entry = ct->manager.find<PermutatePlan>(key); entry) {
//    return entry->plan.ptr;
//  }
//
//  BoundedArray<i64> a_dims(src_dims, src_dims_len, true);
//  BoundedArray<i32> a_syms(src_syms, src_dims_len, true);
//  BoundedArray<i64> b_dims(dst_dims, src_dims_len, true);
//  BoundedArray<i32> b_syms(dst_syms, src_dims_len, true);
//  
//  hiptensorTensorDescriptor_t x_desc;
//  CUTENSOR_ASSERT(hiptensorCreateTensorDescriptor(
//              ct->handle,
//              &x_desc,
//              a_dims.size,
//              a_dims.data,
//              NULL,/*stride*/
//              data_type, cutensor_alignment));
//  
//  hiptensorTensorDescriptor_t y_desc;
//  CUTENSOR_ASSERT(hiptensorCreateTensorDescriptor(
//              ct->handle,
//              &y_desc,
//              b_dims.size,
//              b_dims.data,
//              NULL,/*stride*/
//              data_type, cutensor_alignment));
//  
//  hiptensorOperationDescriptor_t op_desc;
//  CUTENSOR_ASSERT(hiptensorCreateElementwiseBinary(
//              ct->handle, &op_desc,
//              x_desc, a_syms.data, HIPTENSOR_OP_IDENTITY,
//              y_desc, b_syms.data, HIPTENSOR_OP_IDENTITY,
//              y_desc, b_syms.data,
//              op_type, cutensor_compute_type(id)));
//
//  const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
//  
//  hiptensorPlanPreference_t plan_pref;
//  CUTENSOR_ASSERT(hiptensorCreatePlanPreference(
//              ct->handle,
//              &plan_pref,
//              algo,
//              HIPTENSOR_JIT_MODE_NONE));
//  
//  
//  hiptensorPlan_t plan;
//  CUTENSOR_ASSERT(hiptensorCreatePlan(
//              ct->handle,
//              &plan,
//              op_desc,
//              plan_pref,
//              0 /* not required */));
//  
//  ct->manager.insert(
//    key,
//    BinaryPlan{
//      .plan = plan,
//      .plan_pref = plan_pref,
//      .x_desc = x_desc,
//      .y_desc = y_desc,
//    }
//  );
//
//  return { .ptr = plan, .scratch_len = scratch_len };
//}
  
/////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////

#endif
