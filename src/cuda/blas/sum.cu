#ifndef __BLAS_SUM_ZIG__
#define __BLAS_SUM_ZIG__

#include "blas_utils.cu"

EXTERN_C void reduce_sum(
  dtype id,
  void* handle,
  const void* x,
  void* y,
  len_t n
) {
  const auto _handle = static_cast<hipblasHandle_t>(handle);
  const auto _n = static_cast<int>(n);

  switch (id) {
    case SINGLE: {
      const auto _x = static_cast<const float*>(x);
      const auto _y = static_cast<float*>(y);
      return CUBLAS_ASSERT(hipblasSasum(_handle, _n, _x, 1, _y));
    }
    case DOUBLE: {
      const auto _x = static_cast<const double*>(x);
      const auto _y = static_cast<double*>(y);
      return CUBLAS_ASSERT(hipblasDasum(_handle, _n, _x, 1, _y));
    }
  }
}

#endif
