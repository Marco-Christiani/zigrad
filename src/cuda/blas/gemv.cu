#ifndef __BLAS_GEMV_ZIG__
#define __BLAS_GEMV_ZIG__

#include "blas_utils.cu"

extern "C" void gemv(
    dtype id, 
    void* cublas_handle,
    const void* A, 
    const void* x, 
    void* y, 
    len_t m,
    len_t n,
    bool trans_a,
    double alpha,
    double beta
) {
  const int _m = static_cast<int>(m);
  const int _n = static_cast<int>(n);
  const auto _trans_a = (trans_a) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
  
  switch (id) {
    case SINGLE: {
      const float _alpha = static_cast<float>(alpha);
      const float _beta = static_cast<float>(beta);
      return CUBLAS_ASSERT(hipblasSgemv(
        get_handle(cublas_handle), 
        _trans_a,
        _n, _m,
        &_alpha,
        static_cast<const float*>(A), _n,
        static_cast<const float*>(x), 1,
        &_beta,
        static_cast<float*>(y), 1
      ));
    }
    case DOUBLE: {
      return CUBLAS_ASSERT(hipblasDgemv(
        get_handle(cublas_handle), 
        _trans_a,
        _n, _m,
        &alpha,
        static_cast<const double*>(x), 1,
        static_cast<const double*>(A), _n,
        &beta,
        static_cast<double*>(y), 1
      ));
    }
  }
}

#endif
