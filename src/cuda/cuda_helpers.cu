#ifndef __CUDA_HELPERS_ZIG__
#define __CUDA_HELPERS_ZIG__

#include <cutensor/types.h>
#include <stdio.h>
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "/usr/local/cuda/include/hipblas.h"
#include "cuda_includes.cu"
#include "decls.h"

typedef unsigned char u8;
typedef float f32;
typedef double f64;
typedef int64_t i64;
typedef int32_t i32;
typedef uint64_t u64;
typedef uint32_t u32;

#define WARP_SIZE 32

inline hipStream_t get_stream(void* context) {
  return static_cast<hipStream_t>(context);
}

inline hipblasHandle_t get_handle(void* context) {
  return static_cast<hipblasHandle_t>(context);
}

#define CUDA_ASSERT(err) (HandleCudaError( err, __FILE__, __LINE__ ))
inline void HandleCudaError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUBLAS_ASSERT(err) (handleCublasError( err, __FILE__, __LINE__ ))
inline void handleCublasError(hipblasStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPBLAS_STATUS_SUCCESS) {
      printf("Cublas failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CURESULT_ASSERT(err) (handleCuresultError( err, __FILE__, __LINE__ ))
inline void handleCuresultError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    const char** msg = nullptr;

    hipDrvGetErrorString(err, msg);

    if (*msg) {
      printf("%s in %s at line %d\n", *msg, file, line);
    } else {
      printf("Unkown error in %s at line %d\n", file, line);
    }   
    exit(EXIT_FAILURE);
  }
}

#define CUDNN_ASSERT(err) (handleCudnnError( err, __FILE__, __LINE__ ))
inline void handleCudnnError(hipdnnStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPDNN_STATUS_SUCCESS) {
      printf("CUDNN failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUTENSOR_ASSERT(err) (handleCutensorStatus(err, __FILE__, __LINE__ ))
inline void handleCutensorStatus(hiptensorStatus_t status, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (status != HIPTENSOR_STATUS_SUCCESS) {
      printf("%s in %s at line %d\n", hiptensorGetErrorString(status), file, line);
    exit(EXIT_FAILURE);
  }
}

template <typename T>
T* __alloc_scalar(hipStream_t stream) {
  hipDeviceptr_t dptr;
  const hipStream_t _stream = static_cast<hipStream_t>(stream);
  CURESULT_ASSERT(hipMallocAsync(&dptr, sizeof(T), _stream));
  return reinterpret_cast<T*>(dptr);
}

template <typename T>
inline void __free_scalar(hipStream_t stream, T* s) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(s);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}

template <typename T>
T __transfer_scalar(hipStream_t stream, T* s) {
  T result;
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(s);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(&result, dptr, sizeof(T), _stream));
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
  return result;
}

inline hipStream_t __cublas_stream(void* handle) {
  hipStream_t stream;
  CUBLAS_ASSERT(hipblasGetStream(static_cast<hipblasHandle_t>(handle), &stream));
  return stream;
}

inline hipStream_t __cudnn_stream(void* handle) {
  hipStream_t stream;
  CUDNN_ASSERT(hipdnnGetStream(static_cast<hipdnnHandle_t>(handle), &stream));
  return stream;
}

#define CHECK_INVARIANT(b, msg) (CheckInvariant(b, msg, __FILE__, __LINE__ ))
inline void CheckInvariant(bool check, const char* message, const char *file, int line)
{
  if (!(check)) {
    printf("%s in %s at line %d\n", (message), file, line);
    exit(EXIT_FAILURE);
  }
}

#define SYSTEM_EXIT(msg) (SystemExit(msg, __FILE__, __LINE__ ))
inline void SystemExit(const char* message, const char *file, int line)
{
  printf("%s in %s at line %d\n", (message), file, line);
  exit(EXIT_FAILURE);
}

#endif

