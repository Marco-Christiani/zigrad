#ifndef __CUDA_HELPERS_ZIG__
#define __CUDA_HELPERS_ZIG__

#include <stdio.h>
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "/usr/local/cuda/include/hipblas.h"
#include "cuda_includes.cu"
#include "decls.h"

typedef float f32;
typedef double f64;

inline hipStream_t get_stream(void* context) {
  return static_cast<hipStream_t>(context);
}

inline hipblasHandle_t get_handle(void* context) {
  return static_cast<hipblasHandle_t>(context);
}

#define CUDA_ASSERT(err) (HandleCudaError( err, __FILE__, __LINE__ ))
inline void HandleCudaError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUBLAS_ASSERT(err) (handleCublasError( err, __FILE__, __LINE__ ))
inline void handleCublasError(hipblasStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPBLAS_STATUS_SUCCESS) {
      printf("Cublas failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CURESULT_ASSERT(err) (handleCuresultError( err, __FILE__, __LINE__ ))
inline void handleCuresultError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    const char** msg = nullptr;

    hipDrvGetErrorString(err, msg);

    if (*msg) {
      printf("%s in %s at line %d\n", *msg, file, line);
    } else {
      printf("Unkown error in %s at line %d\n", file, line);
    }   
    exit(EXIT_FAILURE);
  }
}

#define CUDNN_ASSERT(err) (handleCudnnError( err, __FILE__, __LINE__ ))
inline void handleCudnnError(hipdnnStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPDNN_STATUS_SUCCESS) {
      printf("CUDNN failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#endif
