#ifndef __CUDA_HELPERS_ZIG__
#define __CUDA_HELPERS_ZIG__

#include <stdio.h>
#include "/usr/local/cuda/include/hip/hip_runtime.h"
#include "/usr/local/cuda/include/hipblas.h"
#include "cuda_includes.cu"
#include "decls.h"

typedef float f32;
typedef double f64;

inline hipStream_t get_stream(void* context) {
  return static_cast<hipStream_t>(context);
}

inline hipblasHandle_t get_handle(void* context) {
  return static_cast<hipblasHandle_t>(context);
}

#define CUDA_ASSERT(err) (HandleCudaError( err, __FILE__, __LINE__ ))
inline void HandleCudaError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
    exit(EXIT_FAILURE);
  }
}

#define CUBLAS_ASSERT(err) (handleCublasError( err, __FILE__, __LINE__ ))
inline void handleCublasError(hipblasStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPBLAS_STATUS_SUCCESS) {
      printf("Cublas failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

#define CURESULT_ASSERT(err) (handleCuresultError( err, __FILE__, __LINE__ ))
inline void handleCuresultError(hipError_t err, const char *file, int line)
{
  if (err != hipSuccess) {
    const char** msg = nullptr;

    hipDrvGetErrorString(err, msg);

    if (*msg) {
      printf("%s in %s at line %d\n", *msg, file, line);
    } else {
      printf("Unkown error in %s at line %d\n", file, line);
    }   
    exit(EXIT_FAILURE);
  }
}

#define CUDNN_ASSERT(err) (handleCudnnError( err, __FILE__, __LINE__ ))
inline void handleCudnnError(hipdnnStatus_t err, const char *file, int line)
{
  // TODO: Report better cublas errors
  if (err != HIPDNN_STATUS_SUCCESS) {
      printf("CUDNN failure in %s at line %d\n", file, line);
    exit(EXIT_FAILURE);
  }
}

template <typename T>
T* __alloc_scalar(hipStream_t stream) {
  hipDeviceptr_t dptr;
  const hipStream_t _stream = static_cast<hipStream_t>(stream);
  CURESULT_ASSERT(hipMallocAsync(&dptr, sizeof(T), _stream));
  return reinterpret_cast<T*>(dptr);
}

template <typename T>
inline void __free_scalar(hipStream_t stream, T* s) {
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(s);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}

template <typename T>
T __transfer_scalar(hipStream_t stream, T* s) {
  T result;
  hipStream_t _stream = static_cast<hipStream_t>(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(s);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(&result, dptr, sizeof(T), _stream));
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
  return result;
}

inline hipStream_t __cublas_stream(void* handle) {
  hipStream_t stream;
  CUBLAS_ASSERT(hipblasGetStream(static_cast<hipblasHandle_t>(handle), &stream));
  return stream;
}

inline hipStream_t __cudnn_stream(void* handle) {
  hipStream_t stream;
  CUDNN_ASSERT(hipdnnGetStream(static_cast<hipdnnHandle_t>(handle), &stream));
  return stream;
}
