#ifndef __NN_UTILS_ZIG__
#define __NN_UTILS_ZIG__
#include "../nn_conflux.h"
#include "../cuda_helpers.cu"
#define CUDNN_DTYPE(id) ((id) == SINGLE) ? HIPDNN_DATA_FLOAT : HIPDNN_DATA_DOUBLE

inline hipdnnSoftmaxAlgorithm_t SMAX_OP_TYPE(smaxtype op) {
  switch (op) {
    case SMAX_FAST:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_FAST;
    case SMAX_MAX:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_ACCURATE;
    default:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_LOG;
    } 
}
  


#endif
