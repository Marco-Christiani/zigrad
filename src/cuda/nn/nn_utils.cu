#ifndef __NN_UTILS_ZIG__
#define __NN_UTILS_ZIG__
#include "../nn_conflux.h"
#include "../blas_conflux.h"
#include "../cuda_helpers.cu"
#include "../device_properties.cu"
#define CUDNN_DTYPE(id) ((id) == SINGLE) ? HIPDNN_DATA_FLOAT : HIPDNN_DATA_DOUBLE

inline hipdnnSoftmaxAlgorithm_t SMAX_OP_TYPE(smaxtype op) {
  switch (op) {
    case SMAX_FAST:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_FAST;
    case SMAX_MAX:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_ACCURATE;
    default:
      return hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_LOG;
    } 
}
  


#endif
