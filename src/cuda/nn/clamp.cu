#include "hip/hip_runtime.h"
#ifndef __NN_CLAMP_ZIG__
#define __NN_CLAMP_ZIG__

#include "nn_utils.cu"
#include <cmath>

template<typename T>
void __clamp(
  void* stream,
  const void* x,
  void* y,
  len_t n,
  T lower,
  T upper
) {
  const auto _stream = static_cast<hipStream_t>(stream);
  const auto x_iter = static_cast<const T*>(x);
  const auto y_iter = static_cast<T*>(y);
  thrust::transform(
      thrust::cuda::par.on(_stream), 
      x_iter,  
      x_iter + n,
      y_iter,
      [=] __device__ (T a) -> T { return thrust::min(thrust::max(lower, a), upper); }
  );
}

extern "C" void clamp(
  dtype id,
  void* stream,
  const void* x,
  void* y,
  len_t n,
  double lower,
  double upper
) {
  switch (id) {
    case SINGLE: {
      return __clamp<f32>(stream, x, y, n, static_cast<f32>(lower), static_cast<f32>(upper));
    }
    case DOUBLE: {
      return __clamp<f64>(stream, x, y, n, lower, upper);
    }
  }
}

#endif
