#include "hip/hip_runtime.h"
#ifndef __NN_CLIP_NORM_ZIG__
#define __NN_CLIP_NORM_ZIG__

#include "nn_utils.cu"

// TODO: This is a dynamic kernel. Check the performance of this.

template<typename T>
__global__ void __clip_norm_kernel(
  void* x,
  len_t n,
  const void* cur_nrm,
  double max_nrm,
  double delta
) {
  const auto _x = static_cast<T*>(x);
  const auto _cur_nrm = *static_cast<const T*>(cur_nrm);
  const auto _max_nrm = static_cast<T>(max_nrm);

  if (_cur_nrm > _max_nrm) {
    const auto _delta = static_cast<T>(delta);
    const auto scale = _max_nrm / (_cur_nrm + _delta);
    thrust::transform(thrust::device, _x, _x + n, _x, [=](T a) -> T { return a * scale; });
  }
}

extern "C" void clip_norm(
  dtype id,
  void* cublas_handle,
  void* x,
  len_t n,
  void* cur_nrm,
  double max_nrm,
  double delta
) {
  nrm2(id, cublas_handle, x, cur_nrm, n);

  const auto stream = __cublas_stream(cublas_handle);

  if (id == SINGLE) {
    return __clip_norm_kernel<f32><<<1,1,0,stream>>>(x, n, cur_nrm, max_nrm, delta);
  } else {
    return __clip_norm_kernel<f64><<<1,1,0,stream>>>(x, n, cur_nrm, max_nrm, delta);
  }

  CUDA_ASSERT(hipPeekAtLastError());
}

#endif
