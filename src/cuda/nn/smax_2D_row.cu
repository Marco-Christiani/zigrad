#ifndef __NN_SMAX_2D_ROW_ZIG__
#define __NN_SMAX_2D_ROW_ZIG__

#include "nn_utils.cu"

extern "C" void smax_2D_row_forward(
  dtype id,
  void* cudnn_handle,
  const void* x,
  void* y,
  len_t m,
  len_t n
) {
  const auto _cudnn_handle = static_cast<hipdnnHandle_t>(cudnn_handle);
  const int _m = static_cast<int>(m);
  const int _n = static_cast<int>(n);

  hipdnnTensorDescriptor_t desc;
  hipdnnCreateTensorDescriptor(&desc);
  hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, CUDNN_DTYPE(id), _m, 1, 1, _n);

  switch (id) {
    case SINGLE: {
      const float alpha = 1.0f;
      const float beta = 0.0f;
      return CUDNN_ASSERT(hipdnnSoftmaxForward(
        _cudnn_handle,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, x,
        &beta, desc, y
      ));
    }
    case DOUBLE: {
      const double alpha = 1.0;
      const double beta = 0.0;
      return CUDNN_ASSERT(hipdnnSoftmaxForward(
        _cudnn_handle,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, x,
        &beta, desc, y
      ));
    }
  }
}

extern "C" void softmax_2D_row_reverse(
  dtype id,
  void* cudnn_handle,
  const void* y_val,
  const void* y_grd,
  void* x_grd,
  len_t m,
  len_t n
) {
  const auto _cudnn_handle = static_cast<hipdnnHandle_t>(cudnn_handle);
  const int _m = static_cast<int>(m);
  const int _n = static_cast<int>(n);

  hipdnnTensorDescriptor_t desc;
  hipdnnCreateTensorDescriptor(&desc);
  hipdnnSetTensor4dDescriptor(desc, HIPDNN_TENSOR_NCHW, CUDNN_DTYPE(id), _m, 1, 1, _n);

  switch (id) {
    case SINGLE: {
      const float alpha = 1.0f;
      const float beta = 1.0f;
      return CUDNN_ASSERT(hipdnnSoftmaxBackward(
        _cudnn_handle,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, y_val, desc, y_grd,
        &beta, desc, x_grd
      ));
    }
    case DOUBLE: {
      const double alpha = 1.0;
      const double beta = 1.0;
      return CUDNN_ASSERT(hipdnnSoftmaxBackward(
        _cudnn_handle,
        HIPDNN_SOFTMAX_ACCURATE,
        HIPDNN_SOFTMAX_MODE_INSTANCE,
        &alpha, desc, y_val, desc, y_grd,
        &beta, desc, x_grd
      ));
    }
  }
}

#endif
