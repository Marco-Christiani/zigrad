#ifndef __BLAS_GEMM_ZIG__
#define __BLAS_GEMM_ZIG__

#include "blas_utils.cu"

// we're using double because every float can cast
// up to a double and then we can go back down.

extern "C" void gemm(
  dtype id,
  void* cublas_handle,
  const void* a_data,
  const void* b_data,
  void* c_data,
  len_t m, 
  len_t n, 
  len_t k,
  bool trans_a,
  bool trans_b, 
  len_t lda,
  len_t ldb,
  len_t ldc,
  double alpha,
  double beta
) {
  const int _m = static_cast<int>(m);
  const int _n = static_cast<int>(n);
  const int _k = static_cast<int>(k);
  const auto _trans_a = (trans_a) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  const auto _trans_b = (trans_b) ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  
  switch (id) {
    case SINGLE: {
      const float _alpha = static_cast<float>(alpha);
      const float _beta = static_cast<float>(beta);
      return CUBLAS_ASSERT(hipblasSgemm(
          get_handle(cublas_handle), 
          _trans_b,
          _trans_a,
          _k, _m, _n,
          &_alpha, 
          static_cast<const float*>(b_data), ldb,
          static_cast<const float*>(a_data), lda,
          &_beta, 
          static_cast<float*>(c_data), ldc
      ));
    }
    case DOUBLE: {
      return CUBLAS_ASSERT(hipblasDgemm(
          get_handle(cublas_handle), 
          _trans_b,
          _trans_a,
          _k, _m, _n,
          &alpha, 
          static_cast<const double*>(b_data), ldb,
          static_cast<const double*>(a_data), lda,
          &beta, 
          static_cast<double*>(c_data), ldc
      ));
    }
  }
}

#endif
