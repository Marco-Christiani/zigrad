#include <stdio.h>
#include "cuda_utils.h"

#include "cuda_helpers.cu"

extern "C" void* memAlloc(len_t N, void* stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr;
  CURESULT_ASSERT(hipMallocAsync(&dptr, N, _stream));
  return (void*)dptr;
}

extern "C" void memcpyHtoD(void* dev_ptr, const void* cpu_ptr, len_t N, void* stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyHtoDAsync(dptr, cpu_ptr, N, _stream));
}

extern "C" void memcpyDtoH(void* cpu_ptr, void const* dev_ptr, len_t N, void* stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipMemcpyDtoHAsync(cpu_ptr, dptr, N, _stream));
}

extern "C" void memcpyDtoD(void* dst_ptr, void const* src_ptr, len_t N, void* stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dst = reinterpret_cast<hipDeviceptr_t>(dst_ptr);
  hipDeviceptr_t src = reinterpret_cast<hipDeviceptr_t>(src_ptr);
  CURESULT_ASSERT(hipMemcpyDtoDAsync(dst, src, N, _stream));
}

extern "C" void memFree(void* dev_ptr, void* stream) {
  hipStream_t _stream = get_stream(stream);
  hipDeviceptr_t dptr = reinterpret_cast<hipDeviceptr_t>(dev_ptr);
  CURESULT_ASSERT(hipFreeAsync(dptr, _stream));
}

extern "C" void streamSynchronize(void* stream) {
  hipStream_t _stream = get_stream(stream);
  CURESULT_ASSERT(hipStreamSynchronize(_stream));
}

extern "C" void deviceSynchronize() {
  CUDA_ASSERT(hipDeviceSynchronize());
}

extern "C" void initDevice(unsigned device_number) {

    CURESULT_ASSERT(hipInit(device_number));

    hipDevice_t device;
    hipCtx_t context;
    int device_count = 0;

    CURESULT_ASSERT(hipGetDeviceCount(&device_count));

    if (device_count <= device_number) {
        fprintf(stderr, "Error: no devices supporting CUDA\n");
        exit(-1);
    }

    CURESULT_ASSERT(hipDeviceGet(&device, device_number));
    CURESULT_ASSERT(hipCtxCreate(&context, 0, device));
}

// Convenience wrapper for hipGetLastError.
// TODO: make this return values instead of void
extern "C" void checkLastError()
{
  CUDA_ASSERT(hipDeviceSynchronize());
  auto err = hipGetLastError();
  if (err != hipSuccess) {
      fprintf(stderr, "Error %s: %s", hipGetErrorName(err), hipGetErrorString(err));
  }
}

extern "C" len_t deviceTotalMemory(unsigned device) {
  len_t total;
  CURESULT_ASSERT(hipDeviceTotalMem(&total, device));
  return total;
}

extern "C" void* initStream() {
  hipStream_t hip_stream = nullptr;

  // REMINDER: For multi-device support, we need to add a call to:
  //    hipError_t hipCtxGetDevice ( hipDevice_t* device ) 

  CURESULT_ASSERT(hipStreamCreateWithFlags(&hip_stream, hipStreamDefault));
  return reinterpret_cast<void*>(hip_stream);
}

extern "C" void deinitStream(void* stream) {
  // TODO: If devices get set, it's probably a good idea to capture
  //       which device a stream was created on and put that in the
  //       void* object. Research if it's required to deinit
  //       streams on the correct device.
  CURESULT_ASSERT(hipStreamDestroy(get_stream(stream)));
}

extern "C" void* initCublasHandle(void* stream) {
  hipblasHandle_t blas_handle = nullptr;
  CUBLAS_ASSERT(hipblasCreate(&blas_handle));
  CUBLAS_ASSERT(hipblasSetStream(blas_handle, get_stream(stream)));
  return reinterpret_cast<void*>(blas_handle);
}

extern "C" void deinitCublasHandle(void* handle) {
  CUBLAS_ASSERT(hipblasDestroy(get_handle(handle)));
}

extern "C" void* initCudnnHandle(void* stream) {
  hipdnnHandle_t cudnn_handle = nullptr;
  CUDNN_ASSERT(hipdnnCreate(&cudnn_handle));
  CUDNN_ASSERT(hipdnnSetStream(cudnn_handle, get_stream(stream)));
  return reinterpret_cast<void*>(cudnn_handle);
}

extern "C" void deinitCudnnHandle(void* handle) {
  CUDNN_ASSERT(hipdnnDestroy(static_cast<hipdnnHandle_t>(handle)));
}

extern "C" void memFill(dtype id, void* data, len_t n, const void* value, void* stream) {
  
  const auto _stream = static_cast<hipStream_t>(stream);

  switch (id) {
    case SINGLE: {
      const auto iter = static_cast<float*>(data);
      const auto _val = static_cast<const float*>(value);
      return thrust::fill(thrust::cuda::par.on(_stream), iter, iter + n, *_val);
    }
    case DOUBLE: {
      const auto iter = static_cast<double *>(data);
      const auto _val = static_cast<const double*>(value);
      return thrust::fill(thrust::cuda::par.on(_stream), iter, iter + n, *_val);
    }
  }
}

extern "C" void memSequence(dtype id, void* data, len_t n, const void* init, const void* step, void* stream) {
  
  const auto _stream = static_cast<hipStream_t>(stream);

  switch (id) {
    case SINGLE: {
      const auto iter = static_cast<float*>(data);
      const auto _init = static_cast<const float*>(init);
      const auto _step = static_cast<const float*>(step);
      return thrust::sequence(thrust::cuda::par.on(_stream), iter, iter + n, *_init, *_step);
    }
    case DOUBLE: {
      const auto iter = static_cast<double*>(data);
      const auto _init = static_cast<const double*>(init);
      const auto _step = static_cast<const double*>(step);
      return thrust::sequence(thrust::cuda::par.on(_stream), iter, iter + n, *_init, *_step);
    }
  }
}

template <typename T>
struct UniformRandom {
    unsigned seed;
    __host__ __device__
    T operator()(unsigned n) const {
        thrust::default_random_engine rng(this->seed);
        thrust::uniform_real_distribution<T> dist(T(-1), T(1));
        rng.discard(n);
        return dist(rng);
    }
};

template <typename T>
struct NormalRandom {
    unsigned seed;
    __host__ __device__
    T operator()(unsigned n) const {
        thrust::default_random_engine rng(this->seed);
        thrust::normal_distribution<T> dist(T(0), T(1));
        rng.discard(n);
        return dist(rng);
    }
};

template <typename T>
void __mem_random(void* x, len_t n, randtype op, unsigned seed, void* stream) {
  const auto _stream = static_cast<hipStream_t>(stream);
  thrust::counting_iterator<unsigned> idxs(0);
  if (op == UNIFORM) {
    thrust::transform(thrust::cuda::par.on(_stream), idxs, idxs+ n, static_cast<T*>(x), UniformRandom<T>{ .seed = seed });
  } else {
    thrust::transform(thrust::cuda::par.on(_stream), idxs, idxs+ n, static_cast<T*>(x), NormalRandom<T>{ .seed = seed });
  }
}

extern "C" void memRandom(dtype id, void* x, len_t n, randtype op, unsigned seed, void* stream) {
  if (id == SINGLE) {
    return __mem_random<float>(x, n, op, seed, stream);
  } else {
    return __mem_random<double>(x, n, op, seed, stream);
  }
}
